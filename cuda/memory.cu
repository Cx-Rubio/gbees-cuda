// Copyright 2024 by Carlos Rubio, published under BSD 3-Clause License.

#include "memory.h"
#include "macro.h"

// FIXME find state from int to int32_t
// FIXME fix start arrays from 1 and not from 0 ?

/** --- Device global memory allocations --- */

/** Alloc hash-table in device global memory */
void allocHashTableDevice(HashTable* hashTable, Grid* grid){
    hashTable->size = grid->maxCells;
    hashTable->usedSize = 0;
    hashTable->freeSize = 0;
    HANDLE_CUDA( hipMalloc( &hashTable->table, 2 * hashTable->size * sizeof(HashTableEntry) ) );
    HANDLE_CUDA( hipMemset(hashTable->table, 0, 2 * hashTable->size * sizeof(HashTableEntry) ) ); 
    HANDLE_CUDA( hipMalloc( &hashTable->usedList, hashTable->size * sizeof(uint32_t) ) );
    HANDLE_CUDA( hipMalloc( &hashTable->freeList, hashTable->size * sizeof(uint32_t) ) ); // TODO intialize freeList in kernel 1, 2, 3, ...
    HANDLE_CUDA( hipMalloc( &hashTable->heap, hashTable->size * sizeof(Cell) ) );
}

/** --- Device global memory de-allocations --- */

/** Free hash-table in device global memory */
void freeHashTableDevice(HashTable* hashTable){
     HANDLE_CUDA( hipFree( hashTable->table) ); 
     HANDLE_CUDA( hipFree( hashTable->usedList) ); 
     HANDLE_CUDA( hipFree( hashTable->freeList) ); 
     HANDLE_CUDA( hipFree( hashTable->heap) ); 
}


// Compute hash value from the state coordinates TODO
static __device__ uint32_t computeHash(int32_t* state){
    
}

static __device__ int cmpState(int32_t* state1, int32_t* state2){
    return memcmp(state1, state2, sizeof(int32_t) * DIM ) == 0;
}

// Copy hashtable key
static __device__ void copyKey(int32_t* src, int32_t* dst){
    memcpy(dst, src, sizeof(int32_t) * DIM);
}

// Copy Cell
static __device__ void copyCell(Cell* src, Cell* dst){
    memcpy(dst, src, sizeof(Cell));
}

/** Hashtable operations  (device) */

__device__ void insertCell(Cell* cell, , HashTable* hashTable){
    if(hashTable->usedSize >= hashTable.size){
        // TODO launch GRID_FULL_ERROR
        }
    
   uint32_t hash = computeHash(state);   
   uint32_t capacity = 2 * hashTable->size;
   
    for(uint32_t counter = 0; counter <capacity; counter++){
        uint32_t hashIndex = (hash + counter) %= capacity;
        if(!hashTable->table[hashIndex]->usedIndex) ){
            uint32_t usedIndex = hashTable->usedSize;
            hashTable->table[hashIndex]->usedIndex) = usedIndex;
            copyKey(cell->state,  hashTable->table[hashIndex]->key); 
            hashTable->usedList[usedIndex] = hashTable->freeList[ hashTable->freeSize -1 ];
            copyCell(cell, hashTable->usedList[usedIndex])
            hashTable->freeSize--;
            hashTable->usedSize++;            
            return;
            }
    }    
    // TODO launch ILLEGAL_STATE_ERROR   
}

__device__ void deleteCell(Cell* cell, , HashTable* hashTable){
    
}

/** Get cell by grid position (hashcode from the table) */
__device__ Cell* findCell(int32_t* state, HashTable* hashTable){
   uint32_t hash = computeHash(state);   
   uint32_t capacity = 2 * hashTable->size;
   
   for(uint32_t counter = 0; counter <capacity; counter++){
        uint32_t hashIndex = (hash + counter) %= capacity;
        if(equalsState(hashTable->table[hashIndex]->key, state) ){
            uint32_t usedIndex = hashTable->table[hashIndex]->usedIndex;
            uint32_t heapIndex = hashTable->usedList[usedIndex];
            return hashTable->heap + heapIndex;
            }
    }    
    return NULL;  
}

/** Get cell by index from the used list */
__device__ Cell* getCell(uint32_t index, HashTable* hashTable){
    if(index < hashTable->usedSize){
        uint32_t heapIndex = hashTable->usedList[index];
        return hashTable->heap + heapIndex;
    } else {
        return NULL;  
    }
}

