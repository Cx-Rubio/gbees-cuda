#include "hip/hip_runtime.h"
// Copyright 2024 by Carlos Rubio (ULE) and Benjamin Hanson (UCSD), published under BSD 3-Clause License.

#include "kernel.h"
#include <stdio.h>
#include "maths.h"

/** Calculate gaussian probability at state x given mean and covariance */
static __device__ double gaussProbability(int32_t* x, Measurement* measurement);

/** 
 * 
 * 
 * @brief Initialization kernel function 
 * 
 * @param grid the grid object
 */
__global__ void initializationKernel(Grid grid, Measurement* measurements){
    // get used list index
    int usedIndex = (uint32_t)(threadIdx.x + blockIdx.x * blockDim.x);    
    
    if(usedIndex == 0) printf("Kernel usedIndex %d\n", usedIndex); // TODO remove
    
    // check used list size
    if(usedIndex >= grid.usedSize) return;
    
    if(usedIndex > 0 ) return; // FIXME remove this line
    
    // used list entry
    UsedListEntry* usedListEntry = grid.usedList + usedIndex;
    
    // obtain key (state coordinates)
    uint32_t hashtableIndex = usedListEntry->hashTableIndex;
    int32_t* key = grid.table[hashtableIndex].key;
    
    // compute initial probability    
    double prob = gaussProbability(key, measurements);
    
    // update cell
    uint32_t heapIndex = usedListEntry->heapIndex;    
    grid.heap[heapIndex].prob = prob; 

    //if(key[0] == 1 && key[1] == 0 && key[2] == 0) printf("Probability %f\n", prob); // TODO remove
    if(usedIndex == 0) printf("Probability of %d,%d,%d : %f\n", key[0], key[1], key[2], prob); // TODO remove    
}

/** Calculate gaussian probability at state x given mean and covariance */
static __device__ double gaussProbability(int32_t* x, Measurement* measurements){    
    double mInvX[DIM];
    double diff[DIM];
    
    for(int i=0;i<DIM;i++){
        diff[i] = x[i] - measurements[0].mean[i];
    }  
    multiplyMatrixVector( (double*)measurements[0].covInv, diff, mInvX, DIM);
    double dotProduct = computeDotProduct(diff, mInvX, DIM);
    return exp(-0.5 * dotProduct);
    
}

