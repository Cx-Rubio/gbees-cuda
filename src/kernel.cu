#include "hip/hip_runtime.h"
// Copyright 2024 by Carlos Rubio (ULE) and Benjamin Hanson (UCSD), published under BSD 3-Clause License.
#include "config.h"
#include "kernel.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include "maths.h"
#include <float.h>

namespace cg = cooperative_groups;

/** Initialize cells */
static __device__ void initializeCell(uint32_t usedIndex, GridDefinition* gridDefinition, Grid* grid, Model* model, Global* global);

/** Calculate gaussian probability at state x given mean and covariance */
static __device__ double gaussProbability(int32_t* key, GridDefinition* gridDefinition, Measurement* measurements);

/** Initialize advection values */
static __device__ void initializeAdv(GridDefinition* gridDefinition, Model* model, Cell* cell);

/** Initialize ik nodes */
static __device__ void initializeIkNodes(Grid* grid, Cell* cell, uint32_t usedIndex);

/** Initialize boundary value */
static __device__ void initializeBoundary(Cell* cell, Model* model);

/** Initialize Grid boundary */
static __device__ void initializeGridBoundary(int offsetIndex, int iterations, double* localArray, GridDefinition* gridDefinition, Grid* grid, Global* global);

/** Normalize probability distribution */
static __device__ void normalizeDistribution(int offsetIndex, int iterations, double* localArray, double* globalArray, Grid* grid);

/** Compute grid bounds */
static __device__ void gridBounds(double* output, double* localArray, double* globalArray, double boundaryValue, double(*fn)(double, double) );

/** Grow grid */
static __device__ void growGrid(int offsetIndex, int iterations, GridDefinition* gridDefinition, Grid* grid, Model* model);

/** Grow grid from one cell */
static __device__ void growGridFromCell(Cell* cell, GridDefinition* gridDefinition, Grid* grid, Model* model);

/** Grow grid from one cell in one dimension and direction */
static __device__ void growGridDireccional(Cell* cell, int dimension, enum Direction direction, GridDefinition* gridDefinition, Grid* grid, Model* model);

/** Create new cell in the grid */
static __device__ void createCell(int32_t* state, GridDefinition* gridDefinition, Grid* grid, Model* model);


/** Perform the Godunov scheme on the discretized PDF*/
static __global__ void godunov_method(int iterations, Grid* grid, GridDefinition* gridDef);

/** Compute the donor cell upwind value for each grid cell */
static __device__ void update_dcu(Cell* cell, Grid* grid, GridDefinition* gridDef);

/** Compute the corner transport upwind values in each direction */
static __device__ void update_ctu(Cell* cell, Grid* grid, GridDefinition* gridDef);

/** Compute flux from the left */
static __device__ double uPlus(double v);

/** Compute flux from the right */
static __device__ double uMinus(double v);

/** MC flux limiter */
static __device__ double fluxLimiter(double th);


 /** Check CFL condition (computes minimum) */
static __device__ void check_cfl_condition(int offsetIndex, int iterations, double* localArray, double* globalArray, Grid* grid, GridDefinition* gridDef)       

	
/** 
 * @brief This function performs the Godunov scheme on the discretized PDF, which is 2nd-order accurate
          and total variation diminishing
 * 
 * @param cell the cell object
 * @param grid the grid object
 * @param gridDefinition the grid definition
 */
static __global__ void godunov_method(int iterations, Grid* grid, GridDefinition* gridDef)
{
    int offsetIndex = threadIdx.x + blockIdx.x * blockDim.x * iterations;     
    int usedIndex;
    Cell* cell;
    // initialize cells
    for(int iter=0;iter<iterations;iter++){        
      usedIndex = (uint32_t)(offsetIndex + iter * blockDim.x); // index in the used list
      cell = getCell(usedIndex, grid); 
      update_dcu(cell, grid, gridDef);
      update_ctu(cell, grid, gridDef);

    }
}


static __device__ double uPlus(double v){
  return fmax(v, 0.0);
}

static __device__ double uMinus(double v){
  return fmin(v, 0.0);
}

static __device__ double computeFlux(Cell* cell, Cell* iCell, GridDefinition* G, int i)
{
  double F = G->dt*(cell->prob-iCell->prob)/(2*G->dx[i]);
  return F;
}

static __device__ double fluxLimiter(double th)
{
  double min1 = (1.0 + th)/2.0;
  min1 = fmin(min1, 2.0);
  min1 = fmax(min1, 2.0*th);
  return min1;
}

static __device__ void update_dcu(Cell* cell, Grid* grid, GridDefinition* gridDef)
{

  if (cell==NULL){
    return;
  }
  double vUpstream;
  double vDownstream;
  
  cell->dcu = 0;
  Cell* iCell; Cell* kCell;
    for(int i = 0; i < DIM; i++){
        cell->ctu[i] = 0.0;
        iCell = getCell(cell->iNodes[i], grid);
        kCell = getCell(cell->kNodes[i], grid);

        double dcu_p = 0;
        double dcu_m = 0;

	/* Original implementation */
	//vUpstream = iCell->v[i];
	//vDownstream = cell->v[i];

	/*Corrected implementation */
	// Valid only for equispaced meshes
	vUpstream = 0.5*(iCell->v[i] + cell->v[i]);
	vDownstream = 0.5*(cell->v[i] + kCell->v[i]);
	
	
        if(kCell != NULL){
          dcu_p = uPlus(vDownstream) * cell->prob + uMinus(vDownstream) * kCell->prob;
        }else{
          dcu_p = uPlus(vDownstream) * cell->prob;
        }
        if(iCell != NULL){
            dcu_m = uPlus(vUpstream) * iCell->prob + uMinus(vUpstream) * cell->prob;
        }
        cell->dcu -= (gridDef->dt/gridDef->dx[i])*(dcu_p-dcu_m);
    }

}

static __device__ void update_ctu(Cell* cell, Grid* grid, GridDefinition* gridDef)
{
  if (cell == NULL)
    {
      return;
    }
    Cell* iCell;
    Cell* jCell;
    Cell* pCell;
    Cell* iiCell;
    Cell* kCell;
    double th;
    double F;
    
    for(int i = 0; i < DIM; i++){
        iCell = getCell(cell->iNodes[i], grid);
        //TreeNode* i_node = r->i_nodes[i];
        //TreeNode* j_node; TreeNode* p_node;
        if(iCell!=NULL){
          F = computeFlux(cell, iCell, gridDef, i);
	/* Original implementation */
	//vUpstream = iCell->v[i];
	//vDownstream = cell->v[i];

	/*Corrected implementation */
	// Valid only for equispaced meshes
	vUpstream = 0.5*(iCell->v[i] + cell->v[i]);
	vDownstream = 0.5*(cell->v[i] + kCell->v[i]);

	  
            for(int j = 0; j < DIM; j++){

	      vUpstream_j = 0.5*(iCell->v[j] + cell->v[j]);
	      vDownstream = 0.5*(cell->v[i] + kCell->v[i]);

	      
                if (j!=i){
                  jCell = getCell(cell->iNodes[j], grid);
                  pCell = getCell(iCell->iNodes[j], grid);

                  cell->ctu[j]      -= uPlus(vUpstream) * uPlus(cell->v[j]) * F;
                  iCell->ctu[j] -= uMinus(vUpstream) * uMinus(iCell->v[j]) * F;

                  if(jCell!=NULL){
                        jCell->ctu[j] -= uPlus(vUpstream) * uMinus(jCell->v[j]) * F;
                    }
                    if(pCell!=NULL){
                        pCell->ctu[j] -= uMinus(vUpstream) * uMinus(pCell->v[j]) * F;
                    }
                }
            }


	    //High-Resolution Correction Terms

            if (vUpstream>0){
                iiCell = getCell(iCell->iNodes[i], grid);
		if(iiCell != NULL){
                    th = (iCell->prob-iiCell->prob)/(cell->prob-iCell->prob);
                }
		else
		{
                    th = (iCell->prob)/(cell->prob-iCell->prob);
                }
            }else{

	      kCell = getCell(cell->kNodes[i], grid);
                if(kCell != NULL){
                    th = (kCell->prob - cell->prob)/(cell->prob - iCell->prob);
                }else{
                    th = (-cell->prob)/(cell->prob - iCell->prob);
                }
            }

            iCell->ctu[i] += fabs(vUpstream)*(gridDef->dx[i]/gridDef->dt - fabs(vUpstream))*F*fluxLimiter(th);
    
            }
    }


}
	
/** 
 * @brief Initialization kernel function 
 * 
 * @param iterations number of cells that should process the same thread
 * @param gridDefinition the grid definition
 * @param grid the grid object
 * @param model the model
 * @param measurements the list of measurements
 */
__global__ void gbeesKernel(int iterations, GridDefinition gridDefinition, Grid grid, Model model, Global global){
    
    // shared memory for reduction processes
    __shared__ double localArray[THREADS_PER_BLOCK];   
    
    // get used list offset index
    int offsetIndex = threadIdx.x + blockIdx.x * blockDim.x * iterations;     
    
    // initialize cells
    for(int iter=0;iter<iterations;iter++){        
        int usedIndex = (uint32_t)(offsetIndex + iter * blockDim.x); // index in the used list                
        initializeCell(usedIndex, &gridDefinition, &grid, &model, &global); // initialize cell
    }    
    
    // set grid maximum and minimum bounds
    if(model.useBounds){ // TODO test use bounds
        initializeGridBoundary(offsetIndex, iterations, localArray, &gridDefinition, &grid, &global);   
        
        /* if(offsetIndex == 0){
            printf("Bounds min %e\n", gridDefinition.lo_bound);
            printf("Bounds max %e\n", gridDefinition.hi_bound);
        } */
    }
    
    // normalize distribution
    normalizeDistribution(offsetIndex, iterations, localArray, global.reductionArray, &grid);

    //if(key[0] == -3 && key[1] == -2 && key[2] == 5) printf("Probability %e\n", prob);
    //if(usedIndex == 100) printf("Probability of %d,%d,%d : %f\n", key[0], key[1], key[2], prob);
    
    /*if(key[0] == 6 && key[1] == 6 && key[2] == 0){
    if(usedIndex == 0){    
        printf("key %d, %d, %d\n",cell->state[0],cell->state[1],cell->state[2]);
        int dim = 0;
        uint32_t iNode = cell->iNodes[dim];
        uint32_t kNode = cell->kNodes[dim];
        
        if(iNode){
            int heapIndexI = (grid.usedList + (iNode-1))->heapIndex;
            Cell* cellI = &grid.heap[heapIndexI];
            printf("I node %d, %d, %d\n", cellI->state[0], cellI->state[1], cellI->state[2]);
        } else {
            printf("Mo iNode\n");
        }
        
        if(kNode){            
            int heapIndexK = (grid.usedList + (kNode-1))->heapIndex;
            Cell* cellK = &grid.heap[heapIndexK];
            printf("K node %d, %d, %d\n", cellK->state[0], cellK->state[1], cellK->state[2]);   
        } else {
            printf("Mo kNode\n");
        }            
    }*/
    // for each measurement
    for(int nm=0;nm<model.numMeasurements;nm++){
        // select active measurement
        Measurement* measurement = &global.measurements[nm];
        
        // propagate probability distribution until the next measurement
        double mt = 0.0; // time propagated from the last measurement
        //int stepCount = 1; // step count
        while(fabs(mt - measurement->T) > TOL) { 
            growGrid(offsetIndex, iterations, &gridDefinition, &grid, &model);
            
            /*
            check_cfl_condition();
            godunov_method();
            update_prob();
            normalize_tree();
            
            if (step_count % DEL_STEP == 0) { // deletion procedure
                prune_tree();
                normalize_tree(); 
            }
         
            stepCount++;
            */
            // FIXME take account of G.dt
            break; // FIXME remove
        }
        
        // perform Bayesian update for the next measurement
        if(nm < model.numMeasurements -1){
            /*
            meas_up_recursive();
            normalize_tree();
            prune_tree();
            normalize_tree(); 
            */
        }
        break; // FIXME remove
    }
    
    
}

/** Initialize cells */
static __device__ void initializeCell(uint32_t usedIndex, GridDefinition* gridDefinition, Grid* grid, Model* model, Global* global){
    // intialize cells    
    if(usedIndex < grid->usedSize){    
        double prob = 0.0;
        Cell* cell = NULL;
    
        // used list entry
        UsedListEntry* usedListEntry = grid->usedList + usedIndex;
        
        // obtain key (state coordinates)
        uint32_t hashtableIndex = usedListEntry->hashTableIndex;
        int32_t* key = grid->table[hashtableIndex].key;
        
        // compute initial probability    
        prob = gaussProbability(key, gridDefinition, global->measurements);
        
        // update cell          
        cell = getCell(usedIndex, grid);
        cell->new_f = 0;
        
        // compute state
        for(int i=0;i<DIM;i++){
            cell->state[i] = key[i]; // state coordinates
            cell->x[i] = gridDefinition->dx[i] * key[i] + gridDefinition->center[i]; // state value
        }
        
        cell->prob = prob; 
        initializeAdv(gridDefinition, model, cell);
        initializeIkNodes(grid, cell, usedIndex);    
        
        // initialize bounday value
        if(model->useBounds){
            initializeBoundary(cell, model);
        }
    }    
}

/** Calculate gaussian probability at state x given mean and covariance */
static __device__ double gaussProbability(int32_t* key, GridDefinition* gridDefinition, Measurement* measurements){    
    double mInvX[DIM];
    double diff[DIM];
    
    for(int i=0;i<DIM;i++){
        diff[i] = key[i] * gridDefinition->dx[i];
    }
    multiplyMatrixVector( (double*)measurements[0].covInv, diff, mInvX, DIM);
    double dotProduct = computeDotProduct(diff, mInvX, DIM);
    return exp(-0.5 * dotProduct);
}

/** Initialize advection values */
static __device__ void initializeAdv(GridDefinition* gridDefinition, Model* model, Cell* cell){        
    double xk[DIM];
    (*model->callbacks->f)(xk, cell->x, gridDefinition->dx); 

    double sum = 0;
    for(int i = 0; i < DIM; i++){
        cell->v[i] = xk[i];
        sum += fabs(cell->v[i]) / gridDefinition->dx[i];
    }
  
    cell->new_f = 1;
    cell->cfl_dt = 1.0/sum;
    
    /*if(cell->state[0]==0 && cell->state[1]==2 && cell->state[2]==2){
        printf("cell cfl_df %e, v[0] %e, v[1] %e, v[2] %e \n", cell->cfl_dt, cell->v[0], cell->v[1], cell->v[2] );
        } */
}

/**
 * Initialize ik nodes 
 * This function depends on an specific order to fill the usedList ( filled in function initializeHashtable() ).
 */
static __device__ void initializeIkNodes(Grid* grid, Cell* cell, uint32_t usedIndex){        
    uint32_t offset = 1;
    for(int i=DIM-1;;i--){        
        // if is not the first cell in the dimension i        
        if(cell->state[i] > -(int)grid->initialExtent[i]){
            uint32_t iIndex = usedIndex - offset;
            cell->iNodes[i] = iIndex + 1; // reserve 0 for no reference            
        } else {            
            cell->iNodes[i] = 0;
        }
        
        // if is not the last cell in the dimension i        
        if(cell->state[i] < (int)grid->initialExtent[i]){
            uint32_t kIndex = usedIndex + offset;        
            cell->kNodes[i] = kIndex + 1; // reserve 0 for no reference            
        }  else {            
            cell->kNodes[i] = 0;
        }
        
        if(i<=0) break;
        offset *= grid->initialExtent[i] * 2 + 1;
    }    
    cell->ik_f = 1;
}

static __device__ void initializeBoundary(Cell* cell, Model* model){
    double j = (*model->callbacks->j)(cell->x);
    cell->bound_val = j;
}

static __device__ void initializeGridBoundary(int offsetIndex, int iterations, double* localArray, GridDefinition* gridDefinition, Grid* grid, Global* global){
    double boundaryValue = -DBL_MAX;    
    for(int iter=0;iter<iterations;iter++){        
        // index in the used list
        uint32_t usedIndex = (uint32_t)(offsetIndex + iter * blockDim.x);   
        Cell* cell = getCell(usedIndex, grid);        
        if(cell != NULL && cell->bound_val > boundaryValue) boundaryValue = cell->bound_val;
    }
    gridBounds(&gridDefinition->hi_bound, localArray, global->reductionArray, boundaryValue, fmax);
    
    boundaryValue = DBL_MAX;   
    for(int iter=0;iter<iterations;iter++){        
        // index in the used list
        uint32_t usedIndex = (uint32_t)(offsetIndex + iter * blockDim.x);   
        Cell* cell = getCell(usedIndex, grid);        
        if(cell != NULL && cell->bound_val < boundaryValue) boundaryValue = cell->bound_val;
    }
    gridBounds(&gridDefinition->lo_bound, localArray, global->reductionArray, boundaryValue, fmin);
}


/** Normalize probability distribution */
static __device__ void normalizeDistribution(int offsetIndex, int iterations, double* localArray, double* globalArray, Grid* grid){        
    // grid synchronization
    cg::grid_group g = cg::this_grid();      
   
    // store the sum of the cells probability for all the iterations at the local reduction array
    localArray[threadIdx.x] = 0.0;
    for(int iter=0;iter<iterations;iter++){
        uint32_t usedIndex = (uint32_t)(offsetIndex + iter * blockDim.x);           
        Cell* cell = getCell(usedIndex, grid); 
        if(cell != NULL) localArray[threadIdx.x] += cell->prob;
    }
    
    __syncthreads();
    
    // reduction process in shared memory (sequencial addressing)
    for(int s=1;s<blockDim.x;s*=2){
        int indexDst = 2 * s * threadIdx.x;
        int indexSrc = indexDst + s;
        if(indexSrc < blockDim.x){
            localArray[indexDst] += localArray[indexSrc];                        
        }
        __syncthreads();
    }
         
    if(threadIdx.x == 0){        
        // store total sum to global array
        globalArray[blockIdx.x] = localArray[0];       
        
        g.sync();
        
        // reduction process in global memory (sequencial addressing)
        for(int s=1;s<gridDim.x;s*=2){
            int indexDst = 2 * s * blockIdx.x;
            int indexSrc = indexDst + s;
            if(indexSrc < gridDim.x){
                globalArray[indexDst] += globalArray[indexSrc];            
            }
            g.sync();
        }     
    }         
   
    /*if(threadIdx.x == 0){
        printf("prob block sum %e\n", localArray[0]);
        }*/
    
    // at the end, the sum of the probability its at globalArray[0]    
    if(threadIdx.x == 0 && blockIdx.x == 0){
            printf("prob sum %e\n", globalArray[0]);
    }
    
    // update the probability of the cells
    for(int iter=0;iter<iterations;iter++){
        uint32_t usedIndex = (uint32_t)(offsetIndex + iter * blockDim.x);   
        Cell* cell = getCell(usedIndex, grid); 
        if(cell != NULL) cell->prob /= globalArray[0];                    
    }    
}


/** Check CFL condition (computes minimum) */
static __device__ void check_cfl_condition(int offsetIndex, int iterations, double* localArray, double* globalArray, Grid* grid, GridDefinition* gridDef){        
    // grid synchronization
    cg::grid_group g = cg::this_grid();      
   
    // store the sum of the cells probability for all the iterations at the local reduction array
    localArray[threadIdx.x] = 9999.0;
    for(int iter=0;iter<iterations;iter++){
        uint32_t usedIndex = (uint32_t)(offsetIndex + iter * blockDim.x);           
        Cell* cell = getCell(usedIndex, grid); 
        if(cell != NULL) localArray[threadIdx.x] = fmin(localArray[threadIdx.x], cell->cfl_dt);
    }
    
    __syncthreads();
    
    // reduction process in shared memory (sequencial addressing)
    for(int s=1;s<blockDim.x;s*=2){
        int indexDst = 2 * s * threadIdx.x;
        int indexSrc = indexDst + s;
        if(indexSrc < blockDim.x){
	  localArray[indexDst] = fmin(localArray[indexDst],localArray[indexSrc]);                        
        }
        __syncthreads();
    }
         
    if(threadIdx.x == 0){        
        // store total sum to global array
        globalArray[blockIdx.x] = localArray[0];       
        
        g.sync();
        
        // reduction process in global memory (sequencial addressing)
        for(int s=1;s<gridDim.x;s*=2){
            int indexDst = 2 * s * blockIdx.x;
            int indexSrc = indexDst + s;
            if(indexSrc < gridDim.x){
	      globalArray[indexDst] = fmin(globalArray[indexDst],globalArray[indexSrc]);            
            }
            g.sync();
        }     
    }         
   
    /*if(threadIdx.x == 0){
        printf("prob block sum %e\n", localArray[0]);
        }*/
    
    // at the end, the sum of the probability its at globalArray[0]    
    if(threadIdx.x == 0 && blockIdx.x == 0){
            printf("prob sum %e\n", globalArray[0]);
    }

    gridDef->dt = fmin(gridDef->dt, globalArray[0]);

}




/** Set the grid definition bounds with the max and min boundary values of the initial grid cells */
static __device__ void gridBounds(double* output, double* localArray, double* globalArray, double boundaryValue, double(*fn)(double, double) ){
    // grid synchronization
    cg::grid_group g = cg::this_grid();      
    
    // store cell bounday value in the reduction array
    localArray[threadIdx.x] = boundaryValue;
    
    __syncthreads();
    
    // reduction process in shared memory (sequencial addressing)
    for(int s=1;s<blockDim.x;s*=2){
        int indexDst = 2 * s * threadIdx.x;
        int indexSrc = indexDst + s;
        if(indexSrc < blockDim.x){
            localArray[indexDst] = fn(localArray[indexSrc], localArray[indexDst]);                        
        }
        __syncthreads();
    }
        
    if(threadIdx.x == 0){        
        // store total sum to global array
        globalArray[blockIdx.x] = localArray[0];       
        
        g.sync();
        
        // reduction process in global memory (sequencial addressing)   
        for(int s=1;s<gridDim.x;s*=2){
            int indexDst = 2 * s * blockIdx.x;
            int indexSrc = indexDst + s;
            if(indexSrc < gridDim.x){
                globalArray[indexDst] = fn(globalArray[indexSrc], globalArray[indexDst]);            
            }
            g.sync();
        } 
        if(blockIdx.x == 0){
            *output = globalArray[0];
        }
        g.sync();
    }        
}

/** Grow grid */
static __device__ void growGrid(int offsetIndex, int iterations, GridDefinition* gridDefinition, Grid* grid, Model* model){
    for(int iter=0;iter<iterations;iter++){ 
        int usedIndex = (uint32_t)(offsetIndex + iter * blockDim.x); // index in the used list  
        if(usedIndex < grid->usedSize){ 
            Cell* cell = getCell(usedIndex, grid);
            if(cell->prob >= gridDefinition->threshold){
                growGridFromCell(cell, gridDefinition, grid, model);
            }
        }        
    }

    // TODO update ik nodes (at the end of the inserts)    
}

/** Grow grid from one cell */
static __device__ void growGridFromCell(Cell* cell, GridDefinition* gridDefinition, Grid* grid, Model* model){
    for(int dimension=0;dimension<DIM;dimension++){
        if(cell->v[dimension] > 0.0){
            growGridDireccional(cell, dimension, FORWARD, gridDefinition, grid, model);        
        } else if(cell->v[dimension] < 0.0){
            growGridDireccional(cell, dimension, BACKWARD, gridDefinition, grid, model);    
        }
    }
}

/** Grow grid from one cell in one dimension and direction */
static __device__ void growGridDireccional(Cell* cell, int dimension, enum Direction direction, GridDefinition* gridDefinition, Grid* grid, Model* model){
    // check if already exists next face
    uint32_t nextFaceIndex = 0; // initialized to null reference
    int32_t state[DIM]; // state indexes for the new cells
    if(direction == FORWARD) nextFaceIndex = cell->kNodes[dimension];
    else nextFaceIndex = cell->iNodes[dimension];
    
    // create next face if not exists
    if(!nextFaceIndex){
        // create new cell key[dimension] = cell->key[dimension]+direction
        copyKey(cell->state, state);
        state[dimension] += direction;
        createCell(state, gridDefinition, grid, model);
    }
    
    // check edges
    for (int j = 0; j < DIM; j++){
        if(j != dimension){
            if(cell->v[j] > 0.0){
                // create new cell key[dimension] = cell->key[dimension] = key[dimension]+direction & cell->key[j] = cell->key[j]+1
                copyKey(cell->state, state);
                state[dimension] += direction;
                state[j] +=1;
                createCell(state, gridDefinition, grid, model);
            } else if(cell->v[j] < 0.0){
                // create new cell key[dimension] = cell->key[dimension] = key[dimension]+direction & cell->key[j] = cell->key[j]-1
                copyKey(cell->state, state);
                state[dimension] += direction;
                state[j] -=1;
                createCell(state, gridDefinition, grid, model);
            }
        }
    }    
}

/** Create new cell in the grid */
static __device__ void createCell(int32_t* state, GridDefinition* gridDefinition, Grid* grid, Model* model){
    Cell cell;
    
    // compute state
    for(int i=0;i<DIM;i++){
        cell.state[i] = state[i]; // state coordinates
        cell.x[i] = gridDefinition->dx[i] * state[i] + gridDefinition->center[i]; // state value
    }
        
    cell.prob = 0.0; 
    cell.new_f = 0;
    cell.ik_f = 0;
    initializeAdv(gridDefinition, model, &cell);
    // TODO initialize ctu[] y dcu
    
    // FIXME shyncro, critical region
    insertCell(&cell, grid);
    
}
